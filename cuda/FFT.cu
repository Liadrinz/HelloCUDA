#include "hip/hip_runtime.h"
// 一维FFT
#include "hip/hip_runtime.h"
#include ""
#include "../utils/Complex.cu"
#include <iostream>
#include <string>
#include <stdlib.h>
#include <time.h>
#include <Windows.h>

int GetBits(int n) {
    int bits = 0;
    while (n >>= 1) {
        bits++;
    }
    return bits;
}

__device__ int br(int i, int bits) {
    int r = 0;
    do {
        r += i % 2 << --bits;
    } while (i /= 2);
    return r;
}

__device__ void Bufferfly(Complex *a, Complex *b, Complex factor) {
    Complex a1 = (*a) + factor * (*b);
    Complex b1 = (*a) - factor * (*b);
    *a = a1;
    *b = b1;
}

__global__ void FFT(Complex nums[], Complex result[], int n, int bits) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= n) return;
    for (int i = 2; i < 2 * n; i *= 2) {
        if (tid % i == 0) {
            int k = i;
            if (n - tid < k) k = n - tid;
            for (int j = 0; j < k / 2; ++j) {
                Bufferfly(&nums[br(tid + j, bits)], &nums[br(tid + j + k / 2, bits)], Complex::W(k, j));
            }
        }
        __syncthreads();
    }
    result[tid] = nums[br(tid, bits)];
}

void printSequence(Complex nums[], const int N) {
    printf("[");
    for (int i = 0; i < N; ++i) {
        double real = nums[i].real, imag = nums[i].imag;
        if (imag == 0) printf("%.16f", real);
        else {
            if (imag > 0) printf("%.16f+%.16fi", real, imag);
            else printf("%.16f%.16fi", real, imag);
        }
        if (i != N - 1) printf(", ");
    }
    printf("]\n");
}

int main() {
    srand(time(0));
    const int TPB = 1024;
    const int N = 1024 * 32;
    const int bits = GetBits(N);

    Complex *nums = (Complex*)malloc(sizeof(Complex) * N), *dNums, *dResult;
    for (int i = 0; i < N; ++i) {
        nums[i] = Complex::GetRandomReal();
    }
    printf("Length of Sequence: %d\n", N);
    // printf("Before FFT: \n");
    // printSequence(nums, N);

    float s = GetTickCount();

    hipMalloc((void**)&dNums, sizeof(Complex) * N);
    hipMalloc((void**)&dResult, sizeof(Complex) * N);
    hipMemcpy(dNums, nums, sizeof(Complex) * N, hipMemcpyHostToDevice);
    
    dim3 threadPerBlock(TPB);
    dim3 blockNum((N + threadPerBlock.x - 1) / threadPerBlock.x);
    FFT<<<blockNum, threadPerBlock>>>(dNums, dResult, N, bits);

    hipMemcpy(nums, dResult, sizeof(Complex) * N, hipMemcpyDeviceToHost);

    float cost = GetTickCount() - s;
    // printf("After FFT: \n");
    // printSequence(nums, N);
    printf("Time of Transfromation: %fms", cost);

    free(nums);
    hipFree(dNums);
    hipFree(dResult);
}
